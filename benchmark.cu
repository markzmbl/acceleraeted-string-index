
#include <stdint.h>
#include <stdio.h>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include "kernels.cuh"
#include "helpers.h"
#include "globals.h"
#include "sindex.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hipsolver.h"
#include <assert.h>
#include <limits>


#include "helpers.h"
#include <chrono>








int main() {  
    
    // cuda debug
    hipError_t cudaStat1 = hipSuccess;
    
    ky_t* keys = (ky_t*) malloc(NUMKEYS * KEYLEN);
    read_keys(keys, FILENAME);
    
    // parameters
    const fp_t et = 1e-12;
    const ix_size_t pt = 16;
    ix_size_t fstep = 1'000'000;
    ix_size_t bstep = 100'000;
    const ix_size_t min_size = CUDACORES;

    assert(pt <= KEYLEN);
    assert(bstep <= MINSIZE);
    assert(fstep <= NUMKEYS);

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    // group meta data   
    std::vector<group_t> groups;
    grouping(keys, NUMKEYS, et, pt, fstep, bstep, MINSIZE, groups);
    ix_size_t group_n = groups.size();
    ky_t* pivots;
    assert(hipHostMalloc(&pivots, group_n * sizeof(ky_t)) == hipSuccess);
        
    for (ix_size_t group_i = 0; group_i < group_n; ++group_i) {
        group_t* group = groups.data() + group_i;
        memcpy(*(pivots + group_i), group->pivot, sizeof(ky_t));
    }

    std::vector<group_t> roots;
    ix_size_t root_n;
    if (group_n > 1) {
        fp_t factor = 0.1 * group_n / NUMKEYS;
        grouping(pivots, group_n, et, pt, 10, 5, 5, roots);
    }
    root_n = roots.size();

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
 
    char filename[] = "test.bin";
    //serialize(index, filename);
    //index_t index2 = deserialize(filename);


    index_t index = { root_n, roots.data(), group_n, groups.data(), pivots };
    ch_t key[sizeof(ky_t)] = {'B', 'B', 'B', 'D', 'A', 'B', 'C', 'B', 'A', 'B', 'A', 'A', 'B', 'D', 'A', 'B'};
    for (ix_size_t i = 0; i < NUMKEYS; ++i) {
        ky_t* key = keys + i;
        printf("i: %u, v: %u\n", i, get_position(&index, *key, keys, pivots));
    }


    

    return 0;
}
